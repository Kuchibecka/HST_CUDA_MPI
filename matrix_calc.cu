#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <strings.h>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } //ошибки
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void matrixCalc(int *a, int *b, int *res, int N) {//cчитываем колонку и столбец

    int col = blockIdx.x * blockDim.x + threadIdx.x;//при инициализации создаются сами,номер колонки номер столбца
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < N && row < N / 2) {
        res[row * N + col] = a[row * N + col] * b[row];
    }
}

extern "C" void * launch_counter(int res_buf, int * matrix, int * arr, int * res_arr) {

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int *a, *b, *res;

    size_t bytes = res_buf * res_buf * sizeof(int) / 2;
    size_t bytes_res = res_buf * sizeof(int) / 2;

    printf("INFO: Start counting cuda\n");

    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&a), bytes));//выделение памяти на видеокарте
    gpuErrchk(hipMemcpyAsync(a, matrix, bytes, hipMemcpyHostToDevice));// а это матрица,копирование массива из оперативки в память видеокарты
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&b), bytes_res));
    gpuErrchk(hipMemcpyAsync(b, arr, bytes_res, hipMemcpyHostToDevice));// маленький массив

    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&res), bytes));// выделение памяти для рез. матрицы

    int block_size = 16;
    int grid_size = (res_buf + block_size) / block_size;// непросто выделить память. количестов блоков

    dim3 DimGrid(grid_size, grid_size,1);//создаем переменные грид и блок
    dim3 DimBlock(block_size, block_size,1);

    matrixCalc<<<DimGrid,DimBlock>>>(a, b, res, res_buf);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    printf("INFO: Stop counting cuda\n");

    gpuErrchk(hipMemcpyAsync(res_arr, res, bytes, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree( res ));
    gpuErrchk(hipFree( a ));
    gpuErrchk(hipFree( b ));
}

// nvcc -arch=sm_35 -c matrix_calc.cu -o matrix_calc.o